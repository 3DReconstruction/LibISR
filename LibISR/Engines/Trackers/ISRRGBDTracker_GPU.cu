#include "hip/hip_runtime.h"
#include "ISRRGBDTracker_GPU.h"
#include "ISRRGBDTracker_DA.h"

#include "../../Utils/ISRCUDAUtils.h"

#include "../../../LibISRUtils/IOUtil.h"
#include "../../../ORUtils/CUDADefines.h"


using namespace LibISR::Engine;
using namespace LibISR::Objects;

__global__ void evaluateEnergy_device(float* e_device, Vector4f* ptcloud_ptr, ISRShapeUnion* shapeunion, ISRTrackingState* state, int count);



LibISR::Engine::ISRRGBDTracker_GPU::ISRRGBDTracker_GPU(int nObjs, const Vector2i& imgSize) :ISRRGBDTracker(nObjs, true)
{
	Vector2i gridSize((imgSize.x + 15) / 16, (imgSize.y + 15) / 16);

	int e_size = gridSize.x*gridSize.y;
	int g_size = ATb_Size*e_size;
	int h_size = ATA_size*e_size;

	energy_host = new float[e_size];
	gradient_host = new float[g_size];
	hessian_host = new float[h_size];

	ORcudaSafeCall(hipMalloc((void**)&energy_dvic,sizeof(float)*e_size));
	ORcudaSafeCall(hipMalloc((void**)&gradient_divc, sizeof(float)*g_size));
	ORcudaSafeCall(hipMalloc((void**)&hessian_divc, sizeof(float)*h_size));

}
LibISR::Engine::ISRRGBDTracker_GPU::~ISRRGBDTracker_GPU()
{
	delete[] energy_host;
	delete[] gradient_host;
	delete[] hessian_host;

	ORcudaSafeCall(hipFree(energy_dvic));
	ORcudaSafeCall(hipFree(gradient_divc));
	ORcudaSafeCall(hipFree(hessian_divc));
}



void LibISR::Engine::ISRRGBDTracker_GPU::evaluateEnergy(float *energy, Objects::ISRTrackingState * trackerState)
{
	int count = this->frame->ptCloud->dataSize;

	dim3 blockSize(256, 1);
	dim3 gridSize((int)ceil((float)count / (float)blockSize.x), 1);

	Vector4f* ptcloud_ptr = this->frame->ptCloud->GetData(true);

	ORcudaSafeCall(hipMemset(energy_dvic, 0, sizeof(float)*gridSize.x));

	evaluateEnergy_device <<<gridSize, blockSize >>> (energy_dvic, ptcloud_ptr, shapeUnion, trackerState, count);
	
	ORcudaSafeCall(hipMemcpy(energy_host,energy_dvic,sizeof(float)*gridSize.x,hipMemcpyDeviceToHost));

	float e = 0;
	
	for (int i = 0; i < gridSize.x; i++) e += energy_host[i];
	
	energy[0] = e ;
}

void LibISR::Engine::ISRRGBDTracker_GPU::computeJacobianAndHessian(float *gradient, float *hessian, Objects::ISRTrackingState * trackerState) const
{
	int count = this->frame->ptCloud->dataSize;
	Vector4f* ptcloud_ptr = this->frame->ptCloud->GetData(false);

	int noPara = trackerState->numPoses() * 6;
	int noParaSQ = noPara*noPara;

	float *globalGradient = new float[noPara];
	float *globalHessian = new float[noParaSQ];
	float *jacobian = new float[noPara];

	for (int i = 0; i < noPara; i++) globalGradient[i] = 0.0f;
	for (int i = 0; i < noParaSQ; i++) globalHessian[i] = 0.0f;

	for (int i = 0; i < count; i++)
	{
		if (computePerPixelJacobian(jacobian, ptcloud_ptr[i], shapeUnion, trackerState))
		{
			for (int a = 0, counter = 0; a < noPara; a++)
			{
				globalGradient[a] += jacobian[a];
				for (int b = 0; b < noPara; b++, counter++) globalHessian[counter] += jacobian[a] * jacobian[b];
			}
		}
	}

	for (int r = 0; r < noPara; ++r) gradient[r] = globalGradient[r];
	for (int r = 0; r < noParaSQ; ++r) hessian[r] = globalHessian[r];
}

void LibISR::Engine::ISRRGBDTracker_GPU::lableForegroundPixels(Objects::ISRTrackingState * trackerState)
{
	int count = this->frame->ptCloud->dataSize;
	Vector4f* ptcloud_ptr = this->frame->ptCloud->GetData(false);
	Vector4f* rgbd_ptr = this->frame->currentLevel->rgbd->GetData(false);

	float dt;
	int totalpix = 0;

	for (int i = 0; i < count; i++)
	{
		if (ptcloud_ptr[i].w > 0) // in the bounding box and have depth
		{
			dt = findPerPixelDT(ptcloud_ptr[i], this->shapeUnion, trackerState);
			if (fabs(dt) <= 2) { rgbd_ptr[i].w = HIST_FG_PIXEL; }
			else { rgbd_ptr[i].w = HIST_BG_PIXEL; }
		}
	}
}


__global__ void evaluateEnergy_device(float* e_device, Vector4f* ptcloud_ptr, ISRShapeUnion* shapeunion, ISRTrackingState* state, int count)
{
	int locId_global = threadIdx.x + blockIdx.x * blockDim.x, locId_local = threadIdx.x;

	__shared__ float dim_shared[256];

	dim_shared[locId_local] = 0.0f;

	if (locId_global < count)
	{
		Vector4f inpt = ptcloud_ptr[locId_global];
		if (inpt.w > -1.0f) dim_shared[locId_local] = computePerPixelEnergy(inpt, shapeunion, state);
	}

	{ //reduction for e_device
		__syncthreads();

		if (locId_local < 128) dim_shared[locId_local] += dim_shared[locId_local + 128];
		__syncthreads();
		if (locId_local < 64) dim_shared[locId_local] += dim_shared[locId_local + 64];
		__syncthreads();

		if (locId_local < 32) warpReduce(dim_shared, locId_local);

		if (locId_local == 0) e_device[blockIdx.x] = dim_shared[locId_local];
	}
}
